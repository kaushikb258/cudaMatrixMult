
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>

using namespace std;

#define TILE_WIDTH 2

__global__ void MatrixMult(int m, int n, int k, float *a, float *b, float *c)
{

 int row = threadIdx.y + blockIdx.y*blockDim.y;  
 int col = threadIdx.x + blockIdx.x*blockDim.x;  
 
 if((row < m) && (col < k))
 {
  float temp = 0.0;
  for (int i = 0; i < n; ++i)
  {
   temp += a[row*n+i]*b[col+i*k];
  }
  c[row*k+col] = temp; 
 }

}


// main fn
int main(void)
{
 
 int m = 4;
 int n = 6;
 int k = 7;
   
 float* a = new float[m*n];
 float* b = new float[n*k];
 float* c = new float[m*k];
 float *dev_a, *dev_b, *dev_c;
 
 dim3 dimGrid((k-1)/TILE_WIDTH+1,(m-1)/TILE_WIDTH+1,1);
 dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);


 hipMalloc((void**)&dev_a, m*n*sizeof(float));
 hipMalloc((void**)&dev_b, n*k*sizeof(float));
 hipMalloc((void**)&dev_c, m*k*sizeof(float));

 for (int i=0; i<m*n; i++)
 {
  a[i] = sin((float) i);
 }
 
 for (int i=0; i<n*k; i++)
 {
  b[i] = cos((float) i);
 }


 hipMemcpy(dev_a, a, m*n*sizeof(float), hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, n*k*sizeof(float), hipMemcpyHostToDevice);

 MatrixMult<<<dimGrid,dimBlock>>>(m,n,k,dev_a,dev_b,dev_c);
 
 hipMemcpy(c, dev_c, m*k*sizeof(float), hipMemcpyDeviceToHost);


 cout<<"a matrix: \n";
 for (int i=0; i<m; i++)
 {
  for (int j=0; j<n; j++)
  {
   cout<<a[n*i+j]<<" ";
  }
  cout<<"\n";
 }

 cout<<"b matrix: \n";
 for (int i=0; i<n; i++)
 {
  for (int j=0; j<k; j++)
  {
   cout<<b[k*i+j]<<" ";
  }
  cout<<"\n";
 }

 cout<<"c matrix: \n";
 for (int i=0; i<m; i++)
 {
  for (int j=0; j<k; j++)
  {
   cout<<c[k*i+j]<<" ";
  }
  cout<<"\n";
 }


 hipFree(dev_a);
 hipFree(dev_b);
 hipFree(dev_c);
 
 delete [] a;
 delete [] b;
 delete [] c;

}
